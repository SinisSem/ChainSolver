#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <cusp/csr_matrix.h>
#include <cusp/coo_matrix.h>
#include <cusp/io/matrix_market.h>
#include <cusp/krylov/gmres.h>
#include <cusp/monitor.h>

// -------------------------------------------------------------
// cuDPP -- CUDA Data Parallel Primitives library
// -------------------------------------------------------------
//  $Revision:
//  $Date:
// ------------------------------------------------------------- 
// This source code is distributed under the terms of license.txt 
// in the root directory of this source distribution.
// ------------------------------------------------------------- 

/**
* @file
* pcr_kernel.cu
*
* @brief CUDPP kernel-level PCR tridiagonal solver
*/

/** \addtogroup cudpp_kernel
* @{
*/
/** @name  Parallel cyclic reduction solver (PCR)
* @{
*/

/**
* @brief Parallel cyclic reduction solver (PCR)
*
* This kernel solves a tridiagonal linear system using the PCR algorithm.
*
* @param[out] d_x Solution vector
* @param[in] d_a Lower diagonal
* @param[in] d_b Main diagonal
* @param[in] d_c Upper diagonal
* @param[in] d_d Right hand side
*/

template <class T>
__global__ void pcrKernel(T *d_a, T *d_b, T *d_c, T *d_d, T *d_x)
{
	int thid = threadIdx.x;
	int blid = blockIdx.x;
	int delta = 1;
	const unsigned int systemSize = blockDim.x;
	int iteration = (int)log2(T(systemSize / 2));

	__syncthreads();

	extern __shared__ char shared[];

	T* a = (T*)shared;
	T* b = (T*)&a[systemSize];//
	T* c = (T*)&b[systemSize];
	T* d = (T*)&c[systemSize];
	T* x = (T*)&d[systemSize];

	a[thid] = d_a[thid + blid * systemSize];
	b[thid] = d_b[thid + blid * systemSize];
	c[thid] = d_c[thid + blid * systemSize];
	d[thid] = d_d[thid + blid * systemSize];

	T aNew, bNew, cNew, dNew;

	__syncthreads();

	//parallel cyclic reduction
	for (int j = 0; j <iteration; j++)
	{
		int i = thid;
		if (i < delta)
		{
			T tmp2 = c[i] / b[i + delta];
			bNew = b[i] - a[i + delta] * tmp2;
			dNew = d[i] - d[i + delta] * tmp2;
			aNew = 0;
			cNew = -c[i + delta] * tmp2;
		}
		else
		{
			if ((systemSize - i - 1) < delta)
			{
				T tmp = a[i] / b[i - delta];
				bNew = b[i] - c[i - delta] * tmp;
				dNew = d[i] - d[i - delta] * tmp;
				aNew = -a[i - delta] * tmp;
				cNew = 0;
			}
			else
			{
				T tmp1 = a[i] / b[i - delta];
				T tmp2 = c[i] / b[i + delta];
				bNew = b[i] - c[i - delta] * tmp1 - a[i + delta] * tmp2;
				dNew = d[i] - d[i - delta] * tmp1 - d[i + delta] * tmp2;
				aNew = -a[i - delta] * tmp1;
				cNew = -c[i + delta] * tmp2;
			}
		}

		__syncthreads();

		b[i] = bNew;
		d[i] = dNew;
		a[i] = aNew;
		c[i] = cNew;

		delta *= 2;
		__syncthreads();
	}

	if (thid < delta)
	{
		int addr1 = thid;
		int addr2 = thid + delta;
		T tmp3 = b[addr2] * b[addr1] - c[addr1] * a[addr2];
		x[addr1] = (b[addr2] * d[addr1] - c[addr1] * d[addr2]) / tmp3;
		x[addr2] = (d[addr2] * b[addr1] - d[addr1] * a[addr2]) / tmp3;
	}

	__syncthreads();
	d_x[thid + blid * systemSize] = x[thid];
}

template <class T>
__global__ void pcrKernelBranchFree(T *d_a, T *d_b, T *d_c, T *d_d, T *d_x)
{
	int thid = threadIdx.x;
	int blid = blockIdx.x;
	int delta = 1;
	const unsigned int systemSize = blockDim.x;
	int iteration = (int)log2(T(systemSize / 2));

	__syncthreads();

	extern __shared__ char shared[];

	T* a = (T*)shared;
	T* b = (T*)&a[systemSize + 1];
	T* c = (T*)&b[systemSize + 1];
	T* d = (T*)&c[systemSize + 1];
	T* x = (T*)&d[systemSize + 1];

	a[thid] = d_a[thid + blid * systemSize];
	b[thid] = d_b[thid + blid * systemSize];
	c[thid] = d_c[thid + blid * systemSize];
	d[thid] = d_d[thid + blid * systemSize];

	T aNew, bNew, cNew, dNew;

	__syncthreads();

	//parallel cyclic reduction
	for (int j = 0; j <iteration; j++)
	{
		int i = thid;

		int iRight = i + delta;
		//iRight = iRight%systemSize;
//		if (iRight >= systemSize) iRight = systemSize - 1;
		iRight = (iRight >= systemSize) * (systemSize)+(!(iRight >= systemSize)) * iRight;

		int iLeft = i - delta;
		//iLeft = iLeft%systemSize;
//		if (iLeft < 0) iLeft = 0;
		iLeft = (!(iLeft < 0)) * iLeft;

		T tmp1 = a[i] / b[iLeft];
		T tmp2 = c[i] / b[iRight];

		bNew = b[i] - c[iLeft] * tmp1 - a[iRight] * tmp2;
		dNew = d[i] - d[iLeft] * tmp1 - d[iRight] * tmp2;
		aNew = -a[iLeft] * tmp1;
		cNew = -c[iRight] * tmp2;

		__syncthreads();

		b[i] = bNew;
		d[i] = dNew;
		a[i] = aNew;
		c[i] = cNew;

		delta *= 2;
		__syncthreads();
	}

	if (thid < delta)
	{
		int addr1 = thid;
		int addr2 = thid + delta;
		T tmp3 = b[addr2] * b[addr1] - c[addr1] * a[addr2];
		x[addr1] = (b[addr2] * d[addr1] - c[addr1] * d[addr2]) / tmp3;
		x[addr2] = (d[addr2] * b[addr1] - d[addr1] * a[addr2]) / tmp3;
	}

	__syncthreads();

	d_x[thid + blid * systemSize] = x[thid];
}

/** @} */ // end tridiagonal functions
/** @} */ // end cudpp_kernel
#include "DefaultConfig.h"
#include "CMTask.h"
#include <cusp/dia_matrix.h>
#include <cusp/io/matrix_market.h>


int main()
{
	RandomChainMatrixGenerator::CMatrixGen Generator = CreateDefaultChains();
	vector<RandomChainMatrixGenerator::Chain>  Chains = Generator.GetChains();
	RandomChainMatrixGenerator::TriDiagonal Tridiag = Chains[0].GetChainMatrix();

	int RowsNumber = Chains[0].ChainMatrix.GetRows();
	int ColsNumber = Chains[0].ChainMatrix.GetCols();
	int ElementsNumber = Chains[0].ChainMatrix.val.size();

	cusp::array1d<float, cusp::device_memory> RightVector = Chains[0].RightVector;
	cusp::array1d<float, cusp::device_memory> X(Chains[0].RightVector.size());
	cusp::array1d<float, cusp::device_memory> A = Tridiag.a;
	cusp::array1d<float, cusp::device_memory> B = Tridiag.b;
	cusp::array1d<float, cusp::device_memory> C = Tridiag.c;

	Chains[0].ChainMatrix.WriteMatrixMarketFile("matr.txt");
	RandomChainMatrixGenerator::CMatrixGen::WriteMatrixMarkeFileVector("rv.txt", Chains[0].RightVector);

	int numSystems = 1;
	int systemSize = Chains[0].RightVector.size();


	const unsigned int num_threads_block = systemSize;

	// setup execution parameters
	dim3  grid(numSystems, 1, 1);
	dim3  threads(num_threads_block, 1, 1);
	float* d_a = thrust::raw_pointer_cast(&A[0]);
	float* d_b = thrust::raw_pointer_cast(&B[0]);
	float* d_c = thrust::raw_pointer_cast(&C[0]);
	float* d_d = thrust::raw_pointer_cast(&RightVector[0]);
	float* d_x = thrust::raw_pointer_cast(&X[0]);
	

	pcrKernel<float> <<< grid, threads, (systemSize + 1) * 5 * sizeof(float) >>>(d_a, d_b, d_c, d_d, d_x);


	cusp::io::write_matrix_market_file(X, "xv.txt");
	//pcrKernelBranchFree<<< grid, threads,(systemSize+1)*5*sizeof(T)>>>(d_a, d_b, d_c, d_d, d_x);

	//CMTask Task1;
	//Task1.Chain.val = thrust::raw_pointer_cast(&ChainCSRMatrix.values[0]);
	//Task1.Chain.row_ptr = thrust::raw_pointer_cast(&ChainCSRMatrix.row_offsets[0]);
	//Task1.Chain.col_idx = thrust::raw_pointer_cast(&ChainCSRMatrix.column_indices[0]);

	//Task1.links_num = 1;
	//Task1.RightVector = thrust::raw_pointer_cast(&RightVector[0]);

	//ker <<<1, 5>>>(Task1);
	//pcrKernelBranchFree


	return 0;
}