
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <cusp/csr_matrix.h>
#include <cusp/coo_matrix.h>
#include <cusp/io/matrix_market.h>
#include <cusp/krylov/gmres.h>
#include <cusp/monitor.h>

int main()
{
	cusp::coo_matrix<int, float, cusp::host_memory> A;
	cusp::io::read_matrix_market_file(A, "..\\Tests\\testM_2014.10.20_1,47,7.txt");
	cusp::csr_matrix<int, float, cusp::host_memory> A1 = A;
	cusp::array1d<float, cusp::host_memory> B;
	cusp::io::read_matrix_market_file(B, "..\\Tests\\testV_2014.10.20_1,47,7.txt");
	cusp::array1d<float, cusp::host_memory> X = B;
	cusp::verbose_monitor<float> monitor(B, 100, 1e-6);
	cusp::krylov::gmres(A1, X, B, 50, monitor);
	return 0;
}
// --������� �� 81 ��������!!!